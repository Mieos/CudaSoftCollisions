#include "hip/hip_runtime.h"
#include "csc/MeshStructureCollider.hpp"
#include <iostream>
#include <math.h> 

__device__ bool checkSphereIntersection(float* centerSphereID, size_t* s1, size_t* s2){


   if((centerSphereID[4*(*s1)+3]==0)||(centerSphereID[4*(*s2)+3]==0)){
      printf("WTF\n");
   }

   float distCenter = 
      (centerSphereID[4*(*s1)] - centerSphereID[4*(*s2)])*
      (centerSphereID[4*(*s1)] - centerSphereID[4*(*s2)])+
      (centerSphereID[4*(*s1)+1] - centerSphereID[4*(*s2)+1])*
      (centerSphereID[4*(*s1)+1] - centerSphereID[4*(*s2)+1])+
      (centerSphereID[4*(*s1)+2] - centerSphereID[4*(*s2)+2])*
      (centerSphereID[4*(*s1)+2] - centerSphereID[4*(*s2)+2]);
   

   distCenter=sqrt(distCenter);
   float addRadius = centerSphereID[4*(*s1)+3] + centerSphereID[4*(*s2)+3]; 

   if(distCenter<addRadius){
      return true;
   } else {
      return false;
   }

} 

__device__ bool checkTetraIntersection(float*  dataPointsD, size_t* idArrayD, size_t* s1, size_t* s2){
   return true;
}

__global__ void updateCenterSphere(float*  dataPointsD, size_t* idArrayD, float* centerSphereID, size_t numberTets){

   size_t numTet = blockIdx.x*blockDim.x*blockDim.y +blockDim.x*threadIdx.y+threadIdx.x;

   size_t id1 = idArrayD[4*numTet];
   size_t id2 = idArrayD[4*numTet+1];
   size_t id3 = idArrayD[4*numTet+2];
   size_t id4 = idArrayD[4*numTet+3];

   if(numTet<numberTets){

      //Coordinate
      centerSphereID[4*numTet] = (dataPointsD[3*id1] + dataPointsD[3*id2] + dataPointsD[3*id3] + dataPointsD[3*id4])/4.0;
      centerSphereID[4*numTet+1] = (dataPointsD[3*id1+1] + dataPointsD[3*id2+1] + dataPointsD[3*id3+1] + dataPointsD[3*id4+1])/4.0;
      centerSphereID[4*numTet+2] = (dataPointsD[3*id1+2] + dataPointsD[3*id2+2] + dataPointsD[3*id3+2] + dataPointsD[3*id4+2])/4.0;
   
      //Radius
      float radius = 
         (centerSphereID[4*numTet] - dataPointsD[3*id1])*(centerSphereID[4*numTet] - dataPointsD[3*id1])+
         (centerSphereID[4*numTet+1] - dataPointsD[3*id1+1])*(centerSphereID[4*numTet+1] - dataPointsD[3*id1+1])+
         (centerSphereID[4*numTet+2] - dataPointsD[3*id1+2])*(centerSphereID[4*numTet+2] - dataPointsD[3*id1+2]);
      centerSphereID[4*numTet+3] = sqrt(radius);

   } else {
      //printf ("DEBUG\n");
   }

}

__global__ void checkForIntersection(float*  dataPointsD, size_t* idArrayD, float* centerSphereID, size_t numberTets, bool* intersectionVector){

   size_t numTet = blockIdx.x*blockDim.x*blockDim.y +blockDim.x*threadIdx.y+threadIdx.x;
   intersectionVector[numTet]=false;

   if(numTet<numberTets){
      //First part
      for(size_t k=0; k<numTet-1; k++){
         if(checkSphereIntersection(centerSphereID,&numTet,&k)){
            //printf("DEB : %u\n", numTet);
            if(checkTetraIntersection(dataPointsD,idArrayD,&numTet,&k)){
               intersectionVector[numTet]=true;
               break;
            }
         }
      }
      //Second part
      if(!intersectionVector[numTet]){
         for(size_t k=numTet+1; k<numberTets; k++){ 
            if(checkSphereIntersection(centerSphereID,&numTet,&k)){ 
               if(checkTetraIntersection(dataPointsD,idArrayD,&numTet,&k)){
                  intersectionVector[numTet]=true;
               }
            }
         }
      }
   } else { 
      //printf ("DEBUG\n");
   }

}

MeshStructureCollider::MeshStructureCollider(const cv::Mat & dataMesh, const std::vector<std::vector<size_t> > & tetIdVector, const std::vector<size_t> & associationVectorU) : initialized(false), numPoints(0), verbose(true){

   std::vector<float> array;

   if(dataMesh.type()==CV_32FC1){

      //std::cout << "Type detected : CV_32FC1 : OK" << std::endl;

      this->numPoints=dataMesh.rows;

      if(dataMesh.cols==3){

         this->dataArrayBuff = new float[3*numPoints];
         
         //Copy the points
         for(size_t i=0; i<dataMesh.rows; i++){
            for(size_t j=0; j<3; j++){
               this->dataArrayBuff[3*i+j]=dataMesh.at<float>(i,j);   
            }
         }

         //Copy points to gpu
         size_t size_data = 3*this->numPoints*sizeof(float);
         hipMalloc((void **) &(this->data_d), size_data);
         hipMemcpy(this->data_d, this->dataArrayBuff, size_data, hipMemcpyHostToDevice);

         //Copy the index of the tetrahedrons
         this->numTets=tetIdVector.size();
         size_t* tetVectorPointer = new size_t[4*tetIdVector.size()];

         for(size_t k=0; k<tetIdVector.size(); k++){

            if(tetIdVector.at(k).size()!=4){
               std::cout << "Issue with association vector in collider initialisation, abort.."<< std::endl;
               break;
            }

            tetVectorPointer[4*k] = tetIdVector.at(k).at(0);
            tetVectorPointer[4*k+1] = tetIdVector.at(k).at(1);
            tetVectorPointer[4*k+2] = tetIdVector.at(k).at(2);
            tetVectorPointer[4*k+3] = tetIdVector.at(k).at(3);

         }

         size_t size_tetVector = 4*this->numTets*sizeof(size_t);
         hipMalloc((void **) &(this->tetId_d), size_tetVector);
         hipMemcpy(this->tetId_d, tetVectorPointer, size_tetVector, hipMemcpyHostToDevice);

         //Copy the association vector
         this->associationVector = associationVectorU;

         //Create sphere buffer: (x y z radius)
         size_t size_sphereBuffer = 4*this->numTets*sizeof(float);
         hipMalloc((void **) &(this->sphereBuf_d), size_sphereBuffer);

         //Create collision vector (CPU and GPU)
         this->collideVectorArray = new bool[this->numTets];
         size_t size_collideVectorArray = this->numTets*sizeof(bool); 
         hipMalloc((void **) &(this->collideVectorArray_d), size_collideVectorArray);

         //Update state
         this->initialized=true;

         delete[] tetVectorPointer;

      } else {

         std::cout << "Wrong number of collumns" << std::endl;

      }

   } else { 
      std::cout << "Type not taken into account in MeshStructureCollider, please convert to CV_32FC1" << std::endl;
   }

}

MeshStructureCollider::~MeshStructureCollider(){

   delete[] this->dataArrayBuff;
   delete[] this->collideVectorArray;
   hipFree(this->data_d);
   hipFree(this->tetId_d);
   hipFree(this->sphereBuf_d);

}

bool MeshStructureCollider::isProperlyInitialized(){

   return this->initialized;

}

bool MeshStructureCollider::updatePointsPositions(const cv::Mat & newPositions){

   //Not init
   if(!this->isProperlyInitialized()){
      return false;
   }

   //Bad number of cols
   if(newPositions.cols!=3){
      return false;
   }

   //Test type
   if(newPositions.type()==CV_32FC1){
   
      for(size_t k=0; k<this->associationVector.size(); k++){
         this->dataArrayBuff[3*k]=newPositions.at<float>(associationVector.at(k),0); 
         this->dataArrayBuff[3*k+1]=newPositions.at<float>(associationVector.at(k),1); 
         this->dataArrayBuff[3*k+2]=newPositions.at<float>(associationVector.at(k),2); 
      }

   } else if(newPositions.type()==CV_64FC1){
   
      for(size_t k=0; k<this->associationVector.size(); k++){
         this->dataArrayBuff[3*k]=float(newPositions.at<double>(associationVector.at(k),0)); 
         this->dataArrayBuff[3*k+1]=float(newPositions.at<double>(associationVector.at(k),1)); 
         this->dataArrayBuff[3*k+2]=float(newPositions.at<double>(associationVector.at(k),2)); 
      }

   } 

   //Copy to buffer
   size_t size_data = 3*this->numPoints*sizeof(float);
   hipMemcpy(this->data_d, this->dataArrayBuff, size_data, hipMemcpyHostToDevice);

   return true;

}

bool MeshStructureCollider::collide(std::vector<bool> & collisionList){

   hipDeviceProp_t prop;
   hipGetDeviceProperties(&prop,0);
   size_t sizeBlockToUse = sqrt (prop.maxThreadsDim[0]);
   size_t sizeGridx = ceil(float(this->numTets)/float(sizeBlockToUse*sizeBlockToUse));
   size_t sizeGridy = 1;

   //std::cout << "Tets = " << this->numTets << std::endl;
   //std::cout << "Block size = " << sizeBlockToUse << std::endl;
   //std::cout << "Grid size = " << sizeGridx << std::endl;

   dim3 dimGrid(sizeGridx,sizeGridy);
   dim3 dimBlock(sizeBlockToUse, sizeBlockToUse);

   //Update sphere
   updateCenterSphere<<<dimGrid, dimBlock>>>(data_d, tetId_d, sphereBuf_d, this->numTets);
   hipDeviceSynchronize();

   //Check collision
   checkForIntersection<<<dimGrid, dimBlock>>>(data_d, tetId_d, sphereBuf_d, this->numTets, this->collideVectorArray_d); 
   hipDeviceSynchronize();

   //Get results
   size_t sizeResult = sizeof(bool)*this->numTets;
   hipMemcpy(collideVectorArray, collideVectorArray_d, sizeResult, hipMemcpyDeviceToHost);
   for(size_t k=0; k<this->numTets;k++){
      collisionList.at(k)=collideVectorArray[k];
   }

   return false; 

}
